#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2020 Neka-Nat
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 **/
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/async/copy.h>

#include "cupoch/knn/kdtree_flann.h"
#include "cupoch/geometry/geometry_utils.h"
#include "cupoch/geometry/pointcloud.h"
#include "cupoch/utility/console.h"
#include "cupoch/utility/helper.h"
#include "cupoch/utility/platform.h"
#include "cupoch/utility/range.h"

using namespace cupoch;
using namespace cupoch::geometry;

namespace {

void SelectByIndexImpl(const geometry::PointCloud &src,
                       geometry::PointCloud &dst,
                       const utility::device_vector<size_t> &indices) {
    const bool has_normals = src.HasNormals();
    const bool has_colors = src.HasColors();
    if (has_normals) dst.normals_.resize(indices.size());
    if (has_colors) dst.colors_.resize(indices.size());
    dst.points_.resize(indices.size());
    thrust::gather(utility::exec_policy(utility::GetStream(0)),
                   indices.begin(), indices.end(), src.points_.begin(),
                   dst.points_.begin());
    if (has_normals) {
        thrust::gather(utility::exec_policy(utility::GetStream(1)),
                       indices.begin(), indices.end(), src.normals_.begin(),
                       dst.normals_.begin());
    }
    if (has_colors) {
        thrust::gather(utility::exec_policy(utility::GetStream(2)),
                       indices.begin(), indices.end(), src.colors_.begin(),
                       dst.colors_.begin());
    }
    cudaSafeCall(hipDeviceSynchronize());
}

struct compute_key_functor {
    compute_key_functor(const Eigen::Vector3f &voxel_min_bound,
                        float voxel_size)
        : voxel_min_bound_(voxel_min_bound), voxel_size_(voxel_size){};
    const Eigen::Vector3f voxel_min_bound_;
    const float voxel_size_;
    __device__ Eigen::Vector3i operator()(const Eigen::Vector3f &pt) {
        auto ref_coord = (pt - voxel_min_bound_) / voxel_size_;
        return Eigen::device_vectorize<float, 3, ::floor>(ref_coord)
                .cast<int>();
    }
};

template <int Index, class... Args>
struct normalize_and_divide_tuple_functor
    : public thrust::binary_function<const thrust::tuple<Args...>,
                                     const int,
                                     thrust::tuple<Args...>> {
    __host__ __device__ thrust::tuple<Args...> operator()(
            const thrust::tuple<Args...> &x, const int &y) const {
        thrust::tuple<Args...> ans = x;
        divide_tuple_impl(ans, y,
                          thrust::make_index_sequence<sizeof...(Args)>{});
        thrust::get<Index>(ans).normalize();
        return ans;
    }
};

struct check_distance_threshold_functor {
    check_distance_threshold_functor(float distance_threshold)
        : distance_threshold_(distance_threshold){};
    const float distance_threshold_;
    __device__ bool operator()(thrust::tuple<int, float> x) const {
        const float dist = thrust::get<1>(x);
        return (dist > 0 && dist < distance_threshold_);
    }
};

struct is_valid_index_functor {
    __device__ int operator()(int idx) const {
        return (int)(idx >= 0);
    }
};

}  // namespace

std::shared_ptr<PointCloud> PointCloud::SelectByIndex(
        const utility::device_vector<size_t> &indices, bool invert) const {
    auto output = std::make_shared<PointCloud>();

    if (invert) {
        size_t n_out = points_.size() - indices.size();
        utility::device_vector<size_t> sorted_indices = indices;
        thrust::sort(utility::exec_policy(0), sorted_indices.begin(),
                     sorted_indices.end());
        utility::device_vector<size_t> inv_indices(n_out);
        thrust::set_difference(thrust::make_counting_iterator<size_t>(0),
                               thrust::make_counting_iterator(points_.size()),
                               sorted_indices.begin(), sorted_indices.end(),
                               inv_indices.begin());
        SelectByIndexImpl(*this, *output, inv_indices);
    } else {
        SelectByIndexImpl(*this, *output, indices);
    }
    return output;
}

std::shared_ptr<PointCloud> PointCloud::SelectByMask(
        const utility::device_vector<bool> &mask, bool invert) const {
    auto output = std::make_shared<PointCloud>();
    if (points_.size() != mask.size()) {
        utility::LogError("[SelectByMask] The point size should be equal to the mask size.\n");
        return output;
    }
    const bool has_normals = HasNormals();
    const bool has_colors = HasColors();
    if (has_normals) output->normals_.resize(mask.size());
    if (has_colors) output->colors_.resize(mask.size());
    output->points_.resize(mask.size());
    auto fn = [invert] __device__ (bool flag) { return invert ? !flag : flag;};
    if (has_normals && has_colors) {
        auto begin = make_tuple_begin(output->points_, output->normals_, output->colors_);
        auto end = thrust::copy_if(make_tuple_begin(points_, normals_, colors_),
                make_tuple_end(points_, normals_, colors_),
                mask.begin(), begin, fn);
        resize_all(thrust::distance(begin, end), output->points_, output->normals_, output->colors_);
    } else if (has_colors) {
        auto begin = make_tuple_begin(output->points_, output->colors_);
        auto end = thrust::copy_if(make_tuple_begin(points_, colors_),
                make_tuple_end(points_, colors_),
                mask.begin(), begin, fn);
        resize_all(thrust::distance(begin, end), output->points_, output->colors_);
    } else if (has_normals) {
        auto begin = make_tuple_begin(output->points_, output->normals_);
        auto end = thrust::copy_if(make_tuple_begin(points_, normals_),
                make_tuple_end(points_, normals_),
                mask.begin(), begin, fn);
        resize_all(thrust::distance(begin, end), output->points_, output->normals_);
    } else {
        auto end = thrust::copy_if(points_.begin(), points_.end(),
                mask.begin(), output->points_.begin(), fn);
        output->points_.resize(thrust::distance(output->points_.begin(), end));
    }
    return output;
}

std::shared_ptr<PointCloud> PointCloud::SelectByIndex(
        const std::vector<size_t> &indices, bool invert) const {
    return SelectByIndex(utility::device_vector<size_t>(indices), invert);
}

std::shared_ptr<PointCloud> PointCloud::SelectByMask(
        const std::vector<bool> &mask, bool invert) const {
    return SelectByMask(utility::device_vector<bool>(mask), invert);
}

std::shared_ptr<PointCloud> PointCloud::VoxelDownSample(
        float voxel_size) const {
    auto output = std::make_shared<PointCloud>();
    if (voxel_size <= 0.0) {
        utility::LogWarning("[VoxelDownSample] voxel_size <= 0.\n");
        return output;
    }

    const Eigen::Vector3f voxel_size3 =
            Eigen::Vector3f(voxel_size, voxel_size, voxel_size);
    const Eigen::Vector3f voxel_min_bound = GetMinBound() - voxel_size3 * 0.5;
    const Eigen::Vector3f voxel_max_bound = GetMaxBound() + voxel_size3 * 0.5;

    if (voxel_size * std::numeric_limits<int>::max() <
        (voxel_max_bound - voxel_min_bound).maxCoeff()) {
        utility::LogWarning("[VoxelDownSample] voxel_size is too small.\n");
        return output;
    }

    const int n = points_.size();
    const bool has_normals = HasNormals();
    const bool has_colors = HasColors();
    compute_key_functor ck_func(voxel_min_bound, voxel_size);
    utility::device_vector<Eigen::Vector3i> keys(n);
    thrust::transform(points_.begin(), points_.end(), keys.begin(), ck_func);

    utility::device_vector<Eigen::Vector3f> sorted_points = points_;
    output->points_.resize(n);
    utility::device_vector<int> counts(n);
    thrust::equal_to<Eigen::Vector3i> binary_pred;
    auto runs = [&keys, &binary_pred] (auto&& out_begins, auto&... params) {
        thrust::sort_by_key(utility::exec_policy(0), keys.begin(),
                            keys.end(),
                            make_tuple_begin(params...));
        add_tuple_functor<typename std::remove_reference_t<decltype(params)>::value_type..., int> add_func;
        auto end = thrust::reduce_by_key(
                utility::exec_policy(0), keys.begin(), keys.end(),
                make_tuple_iterator(std::begin(params)...,
                                    thrust::make_constant_iterator(1)),
                thrust::make_discard_iterator(), out_begins, binary_pred, add_func);
        return thrust::distance(out_begins, end.second);
    };
    if (!has_normals && !has_colors) {
        auto begin = make_tuple_begin(output->points_, counts);
        thrust::sort_by_key(
            utility::exec_policy(0), keys.begin(), keys.end(),
            sorted_points.begin());
        add_tuple_functor<Eigen::Vector3f, int> add_func;
        auto end = thrust::reduce_by_key(
            utility::exec_policy(0), keys.begin(), keys.end(),
            make_tuple_iterator(sorted_points.begin(),
                                thrust::make_constant_iterator(1)),
            thrust::make_discard_iterator(), begin, binary_pred, add_func);
        int n_out = thrust::distance(begin, end.second);
        divide_tuple_functor<Eigen::Vector3f> dv_func;
        auto output_begins = make_tuple_begin(output->points_);
        thrust::transform(output_begins, output_begins + n_out, counts.begin(),
                          output_begins, dv_func);
        output->points_.resize(n_out);
    } else if (has_normals && !has_colors) {
        utility::device_vector<Eigen::Vector3f> sorted_normals = normals_;
        output->normals_.resize(n);
        auto begin =
                make_tuple_begin(output->points_, output->normals_, counts);
        int n_out = runs(begin, sorted_points, sorted_normals);
        normalize_and_divide_tuple_functor<1, Eigen::Vector3f, Eigen::Vector3f>
                dv_func;
        auto output_begins =
                make_tuple_begin(output->points_, output->normals_);
        thrust::transform(output_begins, output_begins + n_out, counts.begin(),
                          output_begins, dv_func);
        resize_all(n_out, output->points_, output->normals_);
    } else if (!has_normals && has_colors) {
        utility::device_vector<Eigen::Vector3f> sorted_colors = colors_;
        resize_all(n, output->colors_);
        auto begin = make_tuple_begin(output->points_, output->colors_, counts);
        int n_out = runs(begin, sorted_points, sorted_colors);
        divide_tuple_functor<Eigen::Vector3f, Eigen::Vector3f> dv_func;
        auto output_begins = make_tuple_begin(output->points_, output->colors_);
        thrust::transform(output_begins, output_begins + n_out, counts.begin(),
                          output_begins, dv_func);
        resize_all(n_out, output->points_, output->colors_);
    } else {
        utility::device_vector<Eigen::Vector3f> sorted_normals = normals_;
        utility::device_vector<Eigen::Vector3f> sorted_colors = colors_;
        resize_all(n, output->normals_, output->colors_);
        auto begin = make_tuple_begin(output->points_, output->normals_,
                                      output->colors_, counts);
        int n_out = runs(begin, sorted_points, sorted_normals, sorted_colors);
        normalize_and_divide_tuple_functor<1, Eigen::Vector3f, Eigen::Vector3f,
                                           Eigen::Vector3f>
                dv_func;
        auto output_begins = make_tuple_begin(output->points_, output->normals_,
                                              output->colors_);
        thrust::transform(output_begins, output_begins + n_out, counts.begin(),
                          output_begins, dv_func);
        resize_all(n_out, output->points_, output->normals_, output->colors_);
    }

    utility::LogDebug(
            "Pointcloud down sampled from {:d} points to {:d} points.\n",
            (int)points_.size(), (int)output->points_.size());
    return output;
}

std::shared_ptr<PointCloud> PointCloud::UniformDownSample(
        size_t every_k_points) const {
    const bool has_normals = HasNormals();
    const bool has_colors = HasColors();
    auto output = std::make_shared<PointCloud>();
    if (every_k_points == 0) {
        utility::LogError("[UniformDownSample] Illegal sample rate.");
        return output;
    }
    const int n_out = points_.size() / every_k_points;
    output->points_.resize(n_out);
    if (has_normals) output->normals_.resize(n_out);
    if (has_colors) output->colors_.resize(n_out);
    thrust::system::cuda::unique_eager_event copy_e[3];
    thrust::strided_range<
            utility::device_vector<Eigen::Vector3f>::const_iterator>
            range_points(points_.begin(), points_.end(), every_k_points);
    copy_e[0] = thrust::async::copy(utility::exec_policy(utility::GetStream(0)),
                 range_points.begin(), range_points.end(),
                 output->points_.begin());
    if (has_normals) {
        thrust::strided_range<
                utility::device_vector<Eigen::Vector3f>::const_iterator>
                range_normals(normals_.begin(), normals_.end(), every_k_points);
        copy_e[1] = thrust::async::copy(utility::exec_policy(utility::GetStream(1)),
                     range_normals.begin(), range_normals.end(),
                     output->normals_.begin());
    }
    if (has_colors) {
        thrust::strided_range<
                utility::device_vector<Eigen::Vector3f>::const_iterator>
                range_colors(colors_.begin(), colors_.end(), every_k_points);
        copy_e[2] = thrust::async::copy(utility::exec_policy(utility::GetStream(2)),
                     range_colors.begin(), range_colors.end(),
                     output->colors_.begin());
    }
    copy_e[0].wait();
    if (has_normals) { copy_e[1].wait(); }
    if (has_colors) { copy_e[2].wait(); }
    return output;
}

std::tuple<std::shared_ptr<PointCloud>, utility::device_vector<size_t>>
PointCloud::RemoveRadiusOutliers(size_t nb_points, float search_radius) const {
    if (nb_points < 1 || search_radius <= 0) {
        utility::LogError(
                "[RemoveRadiusOutliers] Illegal input parameters,"
                "number of points and radius must be positive");
    }
    knn::KDTreeFlann kdtree;
    kdtree.SetRawData(ConvertVector3fVectorRef(*this));
    utility::device_vector<int> tmp_indices;
    utility::device_vector<float> dist;
    kdtree.SearchRadius(points_, search_radius, nb_points + 1, tmp_indices,
                        dist);
    const size_t n_pt = points_.size();
    utility::device_vector<size_t> counts(n_pt);
    utility::device_vector<size_t> indices(n_pt);
    thrust::repeated_range<thrust::counting_iterator<size_t>> range(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator(n_pt), nb_points + 1);
    thrust::reduce_by_key(
            utility::exec_policy(0), range.begin(), range.end(),
            thrust::make_transform_iterator(
                    tmp_indices.begin(),
                    is_valid_index_functor()),
            thrust::make_discard_iterator(), counts.begin(),
            thrust::equal_to<size_t>(), thrust::plus<size_t>());
    auto begin = make_tuple_iterator(indices.begin(),
                                     thrust::make_discard_iterator());
    auto end = thrust::copy_if(
            enumerate_begin(counts), enumerate_end(counts), begin,
            [nb_points] __device__(const thrust::tuple<size_t, size_t> &x) {
                return thrust::get<1>(x) > nb_points;
            });
    indices.resize(thrust::distance(begin, end));
    return std::make_tuple(SelectByIndex(indices), indices);
}

std::tuple<std::shared_ptr<PointCloud>, utility::device_vector<size_t>>
PointCloud::RemoveStatisticalOutliers(size_t nb_neighbors,
                                      float std_ratio) const {
    if (nb_neighbors < 1 || std_ratio <= 0) {
        utility::LogError(
                "[RemoveStatisticalOutliers] Illegal input parameters, number "
                "of neighbors and standard deviation ratio must be positive");
    }
    if (points_.empty()) {
        return std::make_tuple(std::make_shared<PointCloud>(),
                               utility::device_vector<size_t>());
    }
    knn::KDTreeFlann kdtree;
    kdtree.SetRawData(ConvertVector3fVectorRef(*this));
    const size_t n_pt = points_.size();
    utility::device_vector<float> avg_distances(n_pt);
    utility::device_vector<size_t> indices(n_pt);
    utility::device_vector<size_t> counts(n_pt);
    utility::device_vector<int> tmp_indices;
    utility::device_vector<float> dist;
    kdtree.SearchKNN(points_, int(nb_neighbors), tmp_indices, dist);
    thrust::repeated_range<thrust::counting_iterator<size_t>> range(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator(n_pt), nb_neighbors);
    thrust::reduce_by_key(
            utility::exec_policy(0), range.begin(), range.end(),
            make_tuple_iterator(thrust::make_constant_iterator<size_t>(1),
                                dist.begin()),
            thrust::make_discard_iterator(),
            make_tuple_iterator(counts.begin(), avg_distances.begin()),
            thrust::equal_to<size_t>(),
            [] __device__(const thrust::tuple<size_t, float> &rhs,
                          const thrust::tuple<size_t, float> &lhs) {
                float rd = thrust::get<1>(rhs);
                size_t rc = thrust::get<0>(rhs);
                if (isinf(rd) || rd < 0.0) {
                    rd = 0.0;
                    rc = 0;
                }
                float ld = thrust::get<1>(lhs);
                size_t lc = thrust::get<0>(lhs);
                if (isinf(ld) || ld < 0.0) {
                    ld = 0.0;
                    lc = 0;
                }
                return thrust::make_tuple(rc + lc, rd + ld);
            });
    thrust::transform(avg_distances.begin(), avg_distances.end(),
                      counts.begin(), avg_distances.begin(),
                      [] __device__(float avg, size_t cnt) {
                          return (cnt > 0) ? avg / (float)cnt : -1.0;
                      });
    auto mean_and_count = thrust::transform_reduce(
            utility::exec_policy(0), avg_distances.begin(),
            avg_distances.end(),
            [] __device__(float const &x) -> thrust::tuple<float, size_t> {
                return thrust::make_tuple(max(x, 0.0f), (size_t)(x >= 0.0));
            },
            thrust::make_tuple(0.0f, size_t(0)),
            add_tuple_functor<float, size_t>());
    const size_t valid_distances = thrust::get<1>(mean_and_count);
    if (valid_distances == 0) {
        return std::make_tuple(std::make_shared<PointCloud>(),
                               utility::device_vector<size_t>());
    }
    float cloud_mean = thrust::get<0>(mean_and_count);
    cloud_mean /= valid_distances;
    const float sq_sum = thrust::transform_reduce(
            utility::exec_policy(0), avg_distances.begin(),
            avg_distances.end(),
            [cloud_mean] __device__(const float x) -> float {
                return (x > 0) ? (x - cloud_mean) * (x - cloud_mean) : 0.0f;
            },
            0.0, thrust::plus<float>());
    // Bessel's correction
    const float std_dev = std::sqrt(sq_sum / (valid_distances - 1));
    const float distance_threshold = cloud_mean + std_ratio * std_dev;
    check_distance_threshold_functor th_func(distance_threshold);
    auto begin = make_tuple_iterator(indices.begin(),
                                     thrust::make_discard_iterator());
    auto end = thrust::copy_if(enumerate_begin(avg_distances),
                               enumerate_end(avg_distances), begin, th_func);
    indices.resize(thrust::distance(begin, end));
    return std::make_tuple(SelectByIndex(indices), indices);
}
