#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2022 Neka-Nat
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 **/
#include <Eigen/Core>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include "cupoch/knn/lbvh_knn.h"

#include <lbvh_index/aabb.cuh>
#include <lbvh_index/lbvh.cuh>
#include <lbvh_index/lbvh_kernels.cuh>
#include <lbvh_index/query_knn_kernels.cuh>

#include "cupoch/knn/kdtree_search_param.h"
#include "cupoch/utility/eigen.h"
#include "cupoch/utility/platform.h"
#include "cupoch/utility/helper.h"


namespace {

template<int Dim>
struct convert_float3_functor {
    convert_float3_functor() {}
    __device__
    float3 operator() (const Eigen::Matrix<float, Dim, 1>& x) {
        return make_float3(x[0], x[1], x[2]);
    }
};


__device__ __host__
lbvh::AABB to_float3_aabb(const cupoch::knn::AABB& aabb) {
    lbvh::AABB aabb_f3;
    aabb_f3.min = make_float3(aabb.first[0], aabb.first[1], aabb.first[2]);
    aabb_f3.max = make_float3(aabb.second[0], aabb.second[1], aabb.second[2]);
    return aabb_f3;
}

template<typename T>
struct convert_float3_and_aabb_functor {
    convert_float3_and_aabb_functor() {}
    __device__
    thrust::tuple<lbvh::AABB, float3> operator() (const T& x) {
        float3 xf3 = make_float3(x[0], x[1], x[2]);
        lbvh::AABB aabb;
        aabb.min = make_float3(x[0], x[1], x[2]);
        aabb.max = make_float3(x[0], x[1], x[2]);
        return thrust::make_tuple(aabb, xf3);
    }
};

}

namespace cupoch {
namespace knn {

LinearBoundingVolumeHierarchyKNN::LinearBoundingVolumeHierarchyKNN(size_t leaf_size, bool compact, bool sort_queries, bool shrink_to_fit)
    : leaf_size_(leaf_size), compact_(compact), sort_queries_(sort_queries), shrink_to_fit_(shrink_to_fit) {
    nodes_ = std::make_unique<utility::device_vector<lbvh::BVHNode>>();
}

LinearBoundingVolumeHierarchyKNN::LinearBoundingVolumeHierarchyKNN(const utility::device_vector<Eigen::Vector3f> &data,
                                                                   size_t leaf_size, bool compact, bool sort_queries, bool shrink_to_fit)
    : leaf_size_(leaf_size), compact_(compact), sort_queries_(sort_queries), shrink_to_fit_(shrink_to_fit) {
    nodes_ = std::make_unique<utility::device_vector<lbvh::BVHNode>>();
    SetRawData(data);
}

LinearBoundingVolumeHierarchyKNN::LinearBoundingVolumeHierarchyKNN(const std::vector<Eigen::Vector3f> &data,
                                                                   size_t leaf_size, bool compact, bool sort_queries, bool shrink_to_fit)
    : leaf_size_(leaf_size), compact_(compact), sort_queries_(sort_queries), shrink_to_fit_(shrink_to_fit) {
    nodes_ = std::make_unique<utility::device_vector<lbvh::BVHNode>>();
    SetRawData(utility::device_vector<Eigen::Vector3f>(data));
}

LinearBoundingVolumeHierarchyKNN::~LinearBoundingVolumeHierarchyKNN() {}

template <typename T>
int LinearBoundingVolumeHierarchyKNN::SearchNN(const utility::device_vector<T> &query,
                                               float radius,
                                               utility::device_vector<unsigned int> &indices,
                                               utility::device_vector<float> &distance2) const{
    if (query.empty() || n_points_ <= 0 || n_nodes_ <= 0)
        return -1;
    T query0 = query[0];
    if (size_t(query0.size()) != dimension_) return -1;
    return SearchNN<typename utility::device_vector<T>::const_iterator,
                    T::RowsAtCompileTime>(query.begin(), query.end(), radius,
                                          indices, distance2);
}

template <typename T>
int LinearBoundingVolumeHierarchyKNN::SearchNN(const T &query,
                           float radius,
                           thrust::host_vector<unsigned int> &indices,
                           thrust::host_vector<float> &distance2) const {
    utility::device_vector<T> query_dv(1, query);
    utility::device_vector<unsigned int> indices_dv;
    utility::device_vector<float> distance2_dv;
    auto result = SearchNN<T>(query_dv, radius, indices_dv, distance2_dv);
    indices = indices_dv;
    distance2 = distance2_dv;
    return result;
}

template <typename T>
int LinearBoundingVolumeHierarchyKNN::SearchNN(const T &query,
                           float radius,
                           std::vector<unsigned int> &indices,
                           std::vector<float> &distance2) const {
    utility::device_vector<T> query_dv(1, query);
    utility::device_vector<unsigned int> indices_dv;
    utility::device_vector<float> distance2_dv;
    auto result = SearchNN<T>(query_dv, radius, indices_dv, distance2_dv);
    indices.resize(indices_dv.size());
    distance2.resize(distance2_dv.size());
    copy_device_to_host(indices_dv, indices);
    copy_device_to_host(distance2_dv, distance2);
    return result;
}

template <typename T>
bool LinearBoundingVolumeHierarchyKNN::SetRawData(const utility::device_vector<T> &data) {
    n_points_ = data.size();
    n_nodes_ = n_points_ * 2 - 1;
    dimension_ = T::SizeAtCompileTime;
    data_float3_.resize(n_points_);
    dim3 block_dim, grid_dim;
    std::tie(block_dim, grid_dim) = utility::SelectBlockGridSizes(n_points_);

    // init aabbs
    utility::device_vector<lbvh::AABB> aabbs(n_points_);
    thrust::transform(data.begin(), data.end(), make_tuple_begin(aabbs, data_float3_), convert_float3_and_aabb_functor<T>());
    T min_data = utility::ComputeMinBound<T::SizeAtCompileTime, typename T::Scalar>(data);
    T max_data = utility::ComputeMaxBound<T::SizeAtCompileTime, typename T::Scalar>(data);
    extent_.first = Eigen::Vector3f(min_data[0], min_data[1], min_data[2]);
    extent_.second = Eigen::Vector3f(max_data[0], max_data[1], max_data[2]);
    auto extent_float3 = to_float3_aabb(extent_);

    // compute the morton codes of the aabbs
    utility::device_vector<lbvh::HashType> morton_codes(n_points_);
    thrust::transform(
        aabbs.begin(), aabbs.end(), morton_codes.begin(),
        [extent_float3] __device__ (const lbvh::AABB& aabb) { return lbvh::morton_code(aabb, extent_float3); });

    // sort everything by the morton codes
    sorted_indices_.resize(morton_codes.size());
    thrust::sequence(sorted_indices_.begin(), sorted_indices_.end());
    thrust::sort_by_key(morton_codes.begin(), morton_codes.end(), make_tuple_begin(sorted_indices_, aabbs));

    // allocate space for the nodes as a raw cuda array
    nodes_->resize(n_nodes_);
    initialize_tree_kernel<<<grid_dim, block_dim>>>(
        thrust::raw_pointer_cast(nodes_->data()), thrust::raw_pointer_cast(aabbs.data()), n_points_);
    cudaSafeCall(hipDeviceSynchronize());
    thrust::device_vector<unsigned int> root_node_index(1, std::numeric_limits<unsigned int>::max());
    construct_tree_kernel<<<grid_dim, block_dim>>>(
        thrust::raw_pointer_cast(nodes_->data()),
        thrust::raw_pointer_cast(root_node_index.data()),
        thrust::raw_pointer_cast(morton_codes.data()), n_points_);
    cudaSafeCall(hipDeviceSynchronize());

    if (leaf_size_ > 1) {
        utility::device_vector<unsigned int> valid(n_nodes_, 1);
        optimize_tree_kernel<<<grid_dim, block_dim>>>(
            thrust::raw_pointer_cast(nodes_->data()),
            thrust::raw_pointer_cast(root_node_index.data()),
            thrust::raw_pointer_cast(valid.data()), leaf_size_, n_points_);
        cudaSafeCall(hipDeviceSynchronize());
        // compact the tree to increase bandwidth
        if (compact_) {
            utility::device_vector<unsigned int> valid_sums(n_nodes_ + 1, 0);
            thrust::inclusive_scan(valid.begin(), valid.end(), valid_sums.begin() + 1);
            int new_node_count = valid_sums[n_nodes_];
            utility::device_vector<unsigned int> valid_sums_aligned(valid_sums.begin(), valid_sums.end() - 1);
            utility::device_vector<unsigned int> isum(n_nodes_);
            thrust::transform(
                enumerate_begin(valid_sums_aligned), enumerate_end(valid_sums_aligned), isum.begin(),
                [] __device__ (const thrust::tuple<unsigned int, unsigned int>& x) { return thrust::get<0>(x) - thrust::get<1>(x); });
            unsigned int free_indices_size = isum[new_node_count];
            utility::device_vector<unsigned int> free(valid_sums);
            free.resize(new_node_count);
            std::tie(block_dim, grid_dim) = utility::SelectBlockGridSizes(new_node_count);
            compute_free_indices_kernel<<<grid_dim, block_dim>>>(
                thrust::raw_pointer_cast(valid_sums.data()), thrust::raw_pointer_cast(isum.data()), thrust::raw_pointer_cast(free.data()), new_node_count);

            unsigned int first_moved = valid_sums[new_node_count];
            std::tie(block_dim, grid_dim) = utility::SelectBlockGridSizes(n_nodes_);
            compact_tree_kernel<<<grid_dim, block_dim>>>(
                thrust::raw_pointer_cast(nodes_->data()),
                thrust::raw_pointer_cast(root_node_index.data()),
                thrust::raw_pointer_cast(valid_sums.data()),
                thrust::raw_pointer_cast(free.data()),
                first_moved, new_node_count, n_nodes_);
            if (shrink_to_fit_) {
                nodes_->resize(new_node_count);
            }
            n_nodes_ = new_node_count;
        }
        root_node_index_ = root_node_index[0];
    }
    return true;
}

template <typename InputIterator, int Dim>
int LinearBoundingVolumeHierarchyKNN::SearchNN(InputIterator first,
                                               InputIterator last,
                                               float radius,
                                               utility::device_vector<unsigned int> &indices,
                                               utility::device_vector<float> &distance2) const {
    size_t num_query = thrust::distance(first, last);
    auto extent_float3 = to_float3_aabb(extent_);
    utility::device_vector<float3> data_float3(num_query);
    thrust::transform(first, last, data_float3.begin(), convert_float3_functor<Dim>());

    utility::device_vector<lbvh::HashType> morton_codes(num_query);
    utility::device_vector<unsigned int> sorted_indices(num_query);
    thrust::sequence(sorted_indices.begin(), sorted_indices.end());
    if (sort_queries_) {
        dim3 block_dim, grid_dim;
        std::tie(block_dim, grid_dim) = utility::SelectBlockGridSizes(num_query);
        compute_morton_points_kernel<<<grid_dim, block_dim>>>(
            thrust::raw_pointer_cast(data_float3.data()), extent_float3, thrust::raw_pointer_cast(morton_codes.data()), num_query);
        cudaSafeCall(hipDeviceSynchronize());
        thrust::sort_by_key(morton_codes.begin(), morton_codes.end(), sorted_indices.begin());
    }

    dim3 block_dim, grid_dim;
    std::tie(block_dim, grid_dim) = utility::SelectBlockGridSizes(num_query);
    indices.resize(num_query, std::numeric_limits<unsigned int>::max());
    distance2.resize(num_query, std::numeric_limits<float>::max());
    utility::device_vector<unsigned int> neighbors(num_query, 0);

    query_knn_kernel<<<grid_dim, block_dim>>>(
        thrust::raw_pointer_cast(nodes_->data()),
        thrust::raw_pointer_cast(data_float3_.data()),
        thrust::raw_pointer_cast(sorted_indices_.data()),
        root_node_index_,
        radius * radius,
        thrust::raw_pointer_cast(data_float3.data()),
        thrust::raw_pointer_cast(sorted_indices.data()),
        num_query,
        thrust::raw_pointer_cast(indices.data()),
        thrust::raw_pointer_cast(distance2.data()),
        thrust::raw_pointer_cast(neighbors.data()));
    cudaSafeCall(hipDeviceSynchronize());
    return 1;
}

template int LinearBoundingVolumeHierarchyKNN::SearchNN<Eigen::Vector3f>(
        const utility::device_vector<Eigen::Vector3f> &query,
        float radius,
        utility::device_vector<unsigned int> &indices,
        utility::device_vector<float> &distance2) const;

template int LinearBoundingVolumeHierarchyKNN::SearchNN<Eigen::Vector3f>(
        const Eigen::Vector3f &query,
        float radius,
        thrust::host_vector<unsigned int> &indices,
        thrust::host_vector<float> &distance2) const;

template int LinearBoundingVolumeHierarchyKNN::SearchNN<Eigen::Vector3f>(
        const Eigen::Vector3f &query,
        float radius,
        std::vector<unsigned int> &indices,
        std::vector<float> &distance2) const;

template bool LinearBoundingVolumeHierarchyKNN::SetRawData<Eigen::Vector3f>(
        const utility::device_vector<Eigen::Vector3f> &data);

}
}